/* C�digo de multiplica��o de Matrizes 
Trabalho 01: Arquitetura de Computadores
discente: Maria da Penha de Andrade Abi Harb
Algoritmo: c�digo CUDA
Ponto flutuante double
*/

#define TILE_WIDTH 16   // Definindo tamanho do ladrilho
#define n  500          // Definindo tamanho da matriz
#define vz  1000        // Definindo quantas vezes ocorre as multiplica��es


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include <time.h>

void __global__ multiplica(double *Md, double *Nd, double *Pd, int Width);

int main(void)
{	
	//vari�veis para contar o tempo
	hipEvent_t start, stop;
	float time;
  
	double *a,*b,*c;
	
	// adicionar declara��o de variaveis vetores das matrizes da GPU
	double *Ga,*Gb,*Gc;
	
	int i;
	
	// vari�veis para threads e blocos
	// importante para configurar a execu��o do processamento e desempenho
	//1024 threads por bloco. Maximo permitido pela configura��o da GPU (tanto device 0 ou 1)
	dim3 blocksize(TILE_WIDTH,TILE_WIDTH); 
	dim3 gridsize(n/TILE_WIDTH,n/TILE_WIDTH);
	
	// Alocacao de memoria para as matrizes a,b,c na CPU (host)
	a=(double *)malloc(n*n*sizeof(double ));
	b=(double *)malloc(n*n*sizeof(double ));
	c=(double *)malloc(n*n*sizeof(double ));
	
	// Alocacao de memoria para as matrizes a,b,c na GPU (device)
	hipMalloc((void **)&Ga,n*n*sizeof(double ));
	hipMalloc((void **)&Gb,n*n*sizeof(double ));
	hipMalloc((void **)&Gc,n*n*sizeof(double ));
	
	// Atribuindo valores para as matrizes a e b
	for (i=0;i<n*n;i++) { 
		a[i]=1.0f;     
		b[i]=0.01f;
		c[i]=0; 
	}
	
	printf("\nTamanho do ladrilho = %d\n",TILE_WIDTH);
	printf("Tamanho da matriz = %d\n",n);
	printf("Numero de vezes = %d\n\n",vz);
	
	// Copiando os dados para a GPU
	hipMemcpy( Ga,a,n*n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( Gb,b,n*n*sizeof(int),hipMemcpyHostToDevice);
	
	//Valor de inicio da contagem do tempo paralelo
	// tamb�m foi testado com a fun��o clock()
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0 );

    //Multiplicar 0, 100 e 1000 vezes
    for(int qt = 0; qt<vz; qt++)
		// Precisa informar o numero e hierarquia de threads
		//Fun��o recebe as vari�veis da GPU - Fun��o Kernel
		multiplica<<<gridsize,blocksize>>>(Ga,Gb,Gc,n);

	//finaliza��o do tempo
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	time/=1000.0;
	printf("Tempo de execucao GPU = %f\n\n",time);
	
	//Copiando os dados para a CPU
	hipMemcpy( c, Gc, n*n*sizeof(double),hipMemcpyDeviceToHost);

	// Liberando memoria
	free(a);    
	free(b);   
	free(c);   
	
	// Liberando memoria da GPU
	hipFree(Ga); 
	hipFree(Gb); 
	hipFree(Gc);
}

//kernel CUDA que ser� rodado na GPU 
void __global__ multiplica(double *Md, double *Nd, double *Pd, int Width)
{
	//memoria compartilhada
	__shared__ double Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ double Nds[TILE_WIDTH][TILE_WIDTH];
	
	//variaveis do Cuda para Threads
	int bx = blockIdx.x;  
	int by = blockIdx.y;
	
	int tx = threadIdx.x; 
	int ty = threadIdx.y;
	
	// Identificando linha e coluna para a matriz Pd trabalhar
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	
	double Pvalue = 0;
	
	// Loop pra preenchimento da matriz multiplica��o
	for (int m = 0; m < Width/TILE_WIDTH; ++m) 
	{
		    // carregando elementos da memoria compartilhada
	        Mds[ty][tx] = Md[Row * Width + (m * TILE_WIDTH + tx)];
	        Nds[ty][tx] = Nd[(m * TILE_WIDTH + ty) * Width + Col];
	        __syncthreads();
	
	        for (int k = 0; k < TILE_WIDTH; ++k)
	            Pvalue += Mds[ty][k] * Nds[k][tx];
	         __syncthreads();
	}
	Pd[Row * Width + Col] = Pvalue;
}


